#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//Funci�n de shift a realizar por el kernel
__global__ void llamadaCelula(int *a, int ladoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila;
	int vecinos_vivos = comprobarVecinos(a, idHilo, ladoMatriz);
	cambiarEstado(a, idHilo, vecinos_vivos);
}
void cambiarEstado(int *a, int idCelula, int contador) {

	int idHilo = idCelula;
	//La celula esta viva 
	if (a[idHilo] = 1) {
		//Hay menos de 2 celulas vivas o mas de 3
		if (contador < 2 || contador > 3) {
			//Matamos la celula
			a[idHilo] = 0;
		}
	}
	//La celula esta viva
	if (a[idHilo] = 0) {
		//Hay 3 celulas vivas alrededor
		if (contador =  3) {
			//La celula nace
			a[idHilo] = 1;
		}
	}
}
int comprobarVecinos(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula, contador = 0;
	hipDeviceSynchronize();
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[3] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
	for (int i = 0; i < 3; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[3] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1] };
	for (int i = 0; i < 3; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[3] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1] };
	for (int i = 0; i < 3; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[3] = { a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
	for (int i = 0; i < 3; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
	for (int i = 0; i < 5; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho (pero no en la esquina) de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
	for (int i = 0; i < 5; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo - ladoMatriz - 1] };
	for (int i = 0; i < 5; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) de la matriz.
	//Creamos array con vecinos de la celula
	int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
	for (int i = 0; i < 5; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
	}
	//Hilo esta en el medio
	//Creamos array con vecinos de la celula
	int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
	for (int i = 0; i < 8; i++) {
		if (vecinos[i] == 1)
		{
			contador = +1;
		}
		
	}
	return contador;
}
int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	printf("Introduzca el tama�o de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	for (int i = 0; i < ladoMatriz*ladoMatriz; i++)
	{
		if (rand() % 2)
		{
			MatrizA[i] = 1;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	//Mostramos los valores de la matriz una vez inicializada.
	printf("Matriz A: \n");
	for (int i = 0; i < ladoMatriz; i++)
	{
		for (int j = 0; j < ladoMatriz; j++)
		{
			printf("%03d ", MatrizA[j + i * ladoMatriz]);
		}
		printf("\n");
	}
	//Realizaci�n de la operaci�n.
	dim3 nBloques(1, 1);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	//Env�o de datos al device.
	int iteracion = 0;
	while (true) {
		iteracion = +1;
		hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, ladoMatriz);
		hipDeviceSynchronize();
		//Env�o de datos al host.
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		//Representaci�n de los resultados.
		printf("Los valores de la matriz en el paso %d:\n", iteracion);
		for (int i = 0; i < ladoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%03d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
	} 
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	free(MatrizA);
}