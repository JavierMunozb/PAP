#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ void comprobarVivo(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	int contadorVivas = 0;
	if (a[idHilo] == 0)
	{
		a[idHilo] = 1;
	}
	else
	{
		a[idHilo] = 0;
	}
}

//Funci�n de comprobaci�n a realizar por el kernel
__global__ void llamadaCelula(int *a, int ladoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	comprobarVivo(a, idHilo, ladoMatriz);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	char caracter = ' ';
	int generacion = 0;
	printf("Introduzca el tama�o de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	for (int i = 0; i < ladoMatriz*ladoMatriz; i++)
	{
		if (rand() % 2)
		{
			MatrizA[i] = 1;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	//Mostramos los valores de la matriz una vez inicializada.
	printf("Matriz A al inicializarse: \n");
	for (int i = 0; i < ladoMatriz; i++)
	{
		for (int j = 0; j < ladoMatriz; j++)
		{
			printf("%d ", MatrizA[j + i * ladoMatriz]);
		}
		printf("\n");
	}
	dim3 nBloques(1, 1);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	//Env�o de datos al device.
	hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
	while (caracter != 'p')
	{
		//Realizaci�n de la operaci�n.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, ladoMatriz);
		//Env�o de datos al host.
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		//Representaci�n de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < ladoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
		caracter = getchar();
		generacion += 1;
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	free(MatrizA);
}