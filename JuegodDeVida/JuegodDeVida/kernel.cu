#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//Funci�n de shift a realizar por el kernel
__global__ void llamadaCelula (int *a, int ladoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	comprobarVivo(a, idHilo, ladoMatriz);
}

int comprobarVivo(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	__syncthreads();
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el borde izquierdo de la matriz.
	if (a[idHilo + 1] == 1 && a[idHilo - 1] == 1 && a[idHilo - ladoMatriz] == 1 && a[idHilo + ladoMatriz] == 0) //Derecha, izquierda y arriba vivas.
	{
		a[idHilo] = 1;
	}
	else if (a[idHilo + 1] == 1 && a[idHilo - 1] == 1 && a[idHilo - ladoMatriz] == 1 && a[idHilo + ladoMatriz] == 0) //Arriba, lateral superior derecha, derecha vivas
	{

	}
	else if (a[idHilo + 1] == 1 && a[idHilo - 1] == 1 && a[idHilo + ladoMatriz] == 1 && a[idHilo - ladoMatriz] == 0) //Derecha, izquierda y abajo vivas.
	{
		a[idHilo] = 1;
	}
	else if (a[id)
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	printf("Introduzca el tama�o de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	for (int i = 0; i < ladoMatriz*ladoMatriz; i++)
	{
		if (rand() % 2)
		{
			MatrizA[i] = 1;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	//Mostramos los valores de la matriz una vez inicializada.
	printf("Matriz A: \n");
	for (int i = 0; i < ladoMatriz; i++)
	{
		for (int j = 0; j < ladoMatriz; j++)
		{
			printf("%03d ", MatrizA[j + i * ladoMatriz]);
		}
		printf("\n");
	}
	//Realizaci�n de la operaci�n.
	dim3 nBloques(1, 1);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	//Env�o de datos al device.
	hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
	shift_matriz << <nBloques, hilosBloque >> > (MatrizA_d, ladoMatriz);
	hipDeviceSynchronize();
	//Env�o de datos al host.
	hipMemcpy(MatrizA, MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
	//Representaci�n de los resultados.
	printf("Los valores de la matriz en el paso %d:\n", i + 1);
	for (int i = 0; i < ladoMatriz; i++)
	{
		for (int j = 0; j < ladoMatriz; j++)
		{
			printf("%03d ", MatrizA[j + i * ladoMatriz]);
		}
		printf("\n");
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	free(MatrizA);
}