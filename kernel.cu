#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

__device__ int comprobarVecinos(int *a, int idCelula, int ladoMatriz, int ladoBloque, int idBloque)
{
	int idHilo = idCelula;
	int bloques = ladoMatriz / ladoBloque; //Numero de bloques en un lado de la matriz
	int contador = 0;
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	if (idBloque == 0)
	{
		//En este bloque solo hay que comprobarel lado izquierdo y el superior.
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda del bloque.
		if (idCelula == 0)
		{
			//Creamos array con vecinos de la celula
			int vecinos[3] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
			for (int i = 0; i < 3; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha del bloque.
		else if (idHilo == ladoBloque - 1)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo - 1],a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1], a[idHilo + ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda del bloque.
		else if (idHilo == (ladoBloque * ladoMatriz - ladoMatriz))
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) del bloque.
		else if (idHilo % ladoMatriz == 0)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) del bloque.
		else if (idHilo < ladoBloque)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	else if (idBloque == bloques - 1)
	{
		//En este bloque solo tenemos que diferenciar los hilo del lado derecho y superior.
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda del bloque.
		if (idCelula == ladoMatriz  - (ladoBloque*(bloques - blockIdx.x)))
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha del bloque.
		else if (idHilo == (ladoMatriz - 1) - (ladoBloque*(bloques - (blockIdx.x + 1))))
		{
			//Creamos array con vecinos de la celula
			int vecinos[3] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la derecha del bloque. (Si no es esquina superior)
		else if (idHilo == (threadIdx.y + 1)* ladoMatriz - 1)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) del bloque.
		else if (idHilo > (ladoMatriz - ladoBloque) &&  idHilo < (ladoMatriz - 1)) // La concicion del > es irrelevante en principio, si esta en este bloque la cumple siempre
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	else if (idBloque == bloques * bloques - bloques) 
	{
		//En este bloque tenemosque comprobar el ladoo izquierdo e inferior.
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda del bloque.
		if (idCelula == ladoMatriz*ladoMatriz - ladoMatriz)
		{
			//Creamos array con vecinos de la celula
			int vecinos[3] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1] };
			for (int i = 0; i < 3; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina inferior) del bloque.
		else if (idHilo % ladoMatriz == 0)
			{
				//Creamos array con vecinos de la celula
				int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
				for (int i = 0; i < 5; i++)
				{
					if (vecinos[i] == 1)
					{
						contador += 1;
					}
				}
		}	
		//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
		else if (idHilo >= ladoMatriz * (ladoMatriz - 1) && idHilo < ladoMatriz * ladoMatriz - (ladoMatriz - ladoBloque))
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo - ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	else if (idBloque == bloques * bloques - 1)
	{
	//En este bloque tenemosque comprobar el ladoo derecho e inferior.
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha del bloque.
		if (idCelula == ladoMatriz * ladoMatriz - 1)
		{
			//Creamos array con vecinos de la celula
			int vecinos[3] = { a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
			for (int i = 0; i < 3; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		else if (idHilo % ladoMatriz == ladoMatriz - 1)
		{
			//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho del bloque.
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		else if (idHilo < ladoMatriz * ladoMatriz - 1  && idHilo > ladoMatriz*ladoMatriz - ladoBloque)
		{
			//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo del bloque.
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo - 1], a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la parte superior de la matriz.
	else if ((idBloque > 0 )&& (idBloque < bloques - 1 ))
	{
		//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba del bloque. (Los unicos especiales del bloque)
		if ((idCelula >= ladoMatriz - (ladoBloque*(bloques - blockIdx.x))) && (idHilo <= (ladoMatriz - 1) - (ladoBloque*(bloques - (blockIdx.x + 1)))))
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1],  a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la parte inferior de la matriz.
	else if ((idBloque > (bloques *(bloques -1))) && (idBloque < (bloques * bloques) - 1))
	{
		//Solo tenemos que comprobar abajo del bloque.
		//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo del bloque. (Los unicos especiales del bloque)
		if ((idCelula >= ladoMatriz* ladoMatriz - (ladoBloque*(bloques - blockIdx.x))) && (idHilo <= ((ladoMatriz * ladoMatriz) - 1) - (ladoBloque*(bloques - (blockIdx.x + 1)))))
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo - 1] , a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la parte derecha de la matriz.
	else if (idBloque % bloques == bloques - 1)
	{
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la derecha del bloque. (Los unicos especiales del bloque)
		if (idHilo % ladoMatriz == ladoMatriz - 1)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Comprobamos si el bloque al que pertenece el hilo que ha llamado al kernel se encuentra en la parte iquierda de la matriz.
	else if (idBloque % bloques == bloques - 1)
	{
		//Comprobamos si el hilo que ha llamado al kernel se encuentra en la derecha del bloque. (Los unicos especiales del bloque)
		if (idHilo % ladoMatriz == 0)
		{
			//Creamos array con vecinos de la celula
			int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
			for (int i = 0; i < 5; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
		//Hilo esta en el medio de la matriz
		else
		{
			//Creamos array con vecinos de la celula
			int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
			for (int i = 0; i < 8; i++)
			{
				if (vecinos[i] == 1)
				{
					contador += 1;
				}
			}
		}
	}
	//Hilo esta en el medio de la matriz
	else
		{
		//Creamos array con vecinos de la celula
		int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 8; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	
	return contador;
}
__device__ void cambiarEstado(int *a, int *aux, int idCelula, int ladoMatriz, int ladoBloque,int idBloque)
{
	int idHilo = idCelula;
	int contador = comprobarVecinos(a, idHilo, ladoMatriz,ladoBloque, idBloque);
	//La celula esta viva 
	if (a[idHilo] == 1 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 celulas vivas o mas de 3
		//Matamos la celula
		aux[idHilo] = 0;
		printf("Celula %d pasa a estar muerta\n", idHilo);
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && contador == 3)
	{
		//Hay 3 celulas vivas alrededor
		//La celula nace
		aux[idHilo] = 1;
		printf("Celula %d pasa a estar viva\n", idHilo);
	}
}
//Funci�n de comprobaci�n a realizar por el kernel
__global__ void llamadaCelula(int *a, int *aux, int ladoMatriz, int ladoBloque)
{
	int idColumna = blockIdx.x*ladoBloque + threadIdx.x;
	int idFila = blockIdx.y*ladoBloque + threadIdx.y;
	int idBloque = blockIdx.x + blockIdx.y*(ladoMatriz/ladoBloque);
	int idHilo = idColumna + idFila *ladoMatriz;
	cambiarEstado(a, aux, idHilo, ladoMatriz,ladoBloque,idBloque);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	int ladoBloque = 8;
	char caracter = ' ';
	int generacion = 0;
	printf("Introduzca el tamano de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	int *MatrizAux_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	hipMalloc((void**)&MatrizAux_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	int contadorSemillas = 0;
	for (int i = 0; i < ladoMatriz * ladoMatriz; i++)
	{

		if (rand() % 100 < 25 && contadorSemillas < 9) //Solo puede haber un maximo de 9 semillas iniciales. Hay una posibilidad del 25% de que la posicion sea semilla.
		{
			MatrizA[i] = 1;
			contadorSemillas++;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	dim3 nBloques(ladoMatriz/ladoBloque, ladoMatriz / ladoBloque);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	caracter = getchar();
	while (caracter != 'p')
	{
		//Representaci�n de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < ladoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
		//Env�o de datos al device.
		hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
		//Realizaci�n de la operaci�n.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, MatrizAux_d, ladoMatriz, ladoBloque);
		//Env�o de datos al host.
		hipMemcpy(MatrizA_d, MatrizAux_d, ladoMatriz * ladoMatriz * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz * ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		caracter = getchar();
		generacion += 1;
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	hipFree(MatrizAux_d);
	free(MatrizA);
}