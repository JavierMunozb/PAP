#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

__device__ int comprobarVecinos(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	int contador = 0;
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	if (idCelula == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	else if (idHilo == ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	else if (idHilo == (ladoMatriz * ladoMatriz - ladoMatriz))
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	else if (idHilo == ladoMatriz * ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) de la matriz.
	else if (idHilo % ladoMatriz == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho (pero no en la esquina) de la matriz.
	else if (idHilo % ladoMatriz == ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
	else if (idHilo >= ladoMatriz * (ladoMatriz - 1) && idHilo < ladoMatriz * ladoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) de la matriz.
	else if (idHilo < ladoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Hilo esta en el medio
	else
	{
		//Creamos array con vecinos de la celula
		int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 8; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	return contador;
}
__device__ void cambiarEstado(int *a, int *aux, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	int contador = comprobarVecinos(a, idHilo, ladoMatriz);
	//La celula esta viva 
	if (a[idHilo] == 1 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 celulas vivas o mas de 3
		//Matamos la celula
		aux[idHilo] = 0;
		printf("Celula %d pasa a estar muerta\n", idHilo);
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && contador == 3)
	{
		//Hay 3 celulas vivas alrededor
		//La celula nace
		aux[idHilo] = 1;
		printf("Celula %d pasa a estar viva\n", idHilo);
	}
}
//Funci�n de comprobaci�n a realizar por el kernel
__global__ void llamadaCelula(int *a, int *aux, int ladoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	cambiarEstado(a, aux, idHilo, ladoMatriz);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	char caracter = ' ';
	int generacion = 0;
	printf("Introduzca el tamano de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	int *MatrizAux_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	hipMalloc((void**)&MatrizAux_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	int contadorSemillas = 0;
	for (int i = 0; i < ladoMatriz * ladoMatriz; i++)
	{
		
		if (rand() % 100 < 25 && contadorSemillas < 9) //Solo puede haber un maximo de 9 semillas iniciales. Hay una posibilidad del 25% de que la posicion sea semilla.
		{
			MatrizA[i] = 1;
			contadorSemillas++;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	dim3 nBloques(1, 1);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	caracter = getchar();
	while (caracter != 'p')
	{
		//Representaci�n de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < ladoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
		//Env�o de datos al device.
		hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
		//Realizaci�n de la operaci�n.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, MatrizAux_d, ladoMatriz);
		//Env�o de datos al host.
		hipMemcpy(MatrizA_d, MatrizAux_d, ladoMatriz * ladoMatriz * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz * ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		caracter = getchar();
		generacion += 1;
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	hipFree(MatrizAux_d);
	free(MatrizA);
}