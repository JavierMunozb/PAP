#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

__device__ int comprobarVecinos(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	int contador = 0;
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	if (idCelula == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	else if (idHilo == ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	else if (idHilo == (ladoMatriz * ladoMatriz - ladoMatriz))
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	else if (idHilo == ladoMatriz * ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) de la matriz.
	else if (idHilo % ladoMatriz == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho (pero no en la esquina) de la matriz.
	else if (idHilo % (ladoMatriz - 1) == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
	else if (idHilo >= ladoMatriz * (ladoMatriz - 1) && idHilo < ladoMatriz * ladoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) de la matriz.
	else if (idHilo < ladoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Hilo esta en el medio
	else
	{
		//Creamos array con vecinos de la celula
		int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 8; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	return contador;
}
__device__ void cambiarEstado(int *a, int idCelula, int ladoMatriz)
{
	int idHilo = idCelula;
	int contador = comprobarVecinos(a, idHilo, ladoMatriz);
	//La celula esta viva 
	if (a[idHilo] == 1 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 celulas vivas o mas de 3
		//Matamos la celula
		a[idHilo] = 0;
		printf("Celula %d pasa a estar muerta\n", idHilo);
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && contador == 3)
	{
		//Hay 3 celulas vivas alrededor
		//La celula nace
		a[idHilo] = 1;
		printf("Celula %d pasa a estar viva\n", idHilo);
	}
}
//Función de comprobación a realizar por el kernel
__global__ void llamadaCelula(int *a, int ladoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	cambiarEstado(a, idHilo, ladoMatriz);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	char caracter;
	int generacion = 0;
	printf("Introduzca el tamano de la matriz. \n");
	scanf("%d", &ladoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*ladoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int));
	//Inicialización de matriz.
	int contadorSemillas = 0;
	for (int i = 0; i < ladoMatriz * ladoMatriz; i++)
	{
		
		if (rand() % 100 < 25)
		{
			MatrizA[i] = 1;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	//Mostramos los valores de la matriz una vez inicializada.
	printf("Matriz A al inicializarse: \n");
	for (int i = 0; i < ladoMatriz; i++)
	{
		for (int j = 0; j < ladoMatriz; j++)
		{
			printf("%d ", MatrizA[j + i * ladoMatriz]);
		}
		printf("\n");
	}
	dim3 nBloques(1, 1);
	dim3 hilosBloque((ladoMatriz + nBloques.x - 1) / nBloques.x, (ladoMatriz + nBloques.y - 1) / nBloques.y);
	//Envío de datos al device.
	hipMemcpy(MatrizA_d, MatrizA, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyHostToDevice);
	while (caracter != 'p')
	{
		//Realización de la operación.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, ladoMatriz);
		//Envío de datos al host.
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz*ladoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		//Representación de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < ladoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
		caracter = getchar();
		generacion += 1;
	}
	//Liberación del espacio usado por los punteros.
	hipFree(MatrizA_d);
	free(MatrizA);
}