#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

__device__ int comprobarVecinos(int *a, int idCelula, int filas, int columnas)
{
	int idHilo = idCelula;
	int contador = 0;
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	if (idCelula == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo + columnas], a[idHilo + columnas + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	else if (idHilo == columnas - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo + columnas], a[idHilo + columnas - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	else if (idHilo == (columnas * filas - columnas))
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo - columnas], a[idHilo - columnas + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	else if (idHilo == columnas * filas - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo - columnas], a[idHilo - columnas - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) de la matriz.
	else if (idHilo % columnas == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo + columnas], a[idHilo - columnas], a[idHilo - columnas + 1], a[idHilo + columnas + 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho (pero no en la esquina) de la matriz.
	else if (idHilo % columnas == columnas - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo - 1], a[idHilo + columnas], a[idHilo - columnas], a[idHilo - columnas - 1], a[idHilo + columnas - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
	else if (idHilo >= columnas * (filas - 1) && idHilo < columnas * filas)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - columnas], a[idHilo - columnas + 1], a[idHilo - columnas - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) de la matriz.
	else if (idHilo < columnas)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + columnas], a[idHilo + columnas + 1], a[idHilo + columnas - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Hilo esta en el medio
	else
	{
		//Creamos array con vecinos de la celula
		int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + columnas], a[idHilo - columnas], a[idHilo - columnas - 1], a[idHilo - columnas + 1], a[idHilo + columnas + 1], a[idHilo + columnas - 1] };
		for (int i = 0; i < 8; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	return contador;
}
__device__ void cambiarEstado(int *a, int *aux, int idCelula, int filas, int columnas)
{
	int idHilo = idCelula;
	int contador = comprobarVecinos(a, idHilo, filas, columnas);
	//La celula esta viva 
	if (a[idHilo] == 1 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 celulas vivas o mas de 3
		//Matamos la celula
		aux[idHilo] = 0;
		printf("Celula %d pasa a estar muerta\n", idHilo);
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && contador == 3)
	{
		//Hay 3 celulas vivas alrededor
		//La celula nace
		aux[idHilo] = 1;
		printf("Celula %d pasa a estar viva\n", idHilo);
	}
}
//Funci�n de comprobaci�n a realizar por el kernel
__global__ void llamadaCelula(int *a, int *aux, int filas, int columnas)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	cambiarEstado(a, aux, idHilo, filas, columnas);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int filas = 0;
	int columnas = 0;
	char caracter = ' ';
	char manual = ' ';
	int generacion = 0;
	printf("Introduzca el numero de filas. \n");
	scanf("%d", &filas);
	printf("Introduzca el numero de columnas. \n");
	scanf("%d", &columnas);
	printf("Introduzca \"m\" si quiere reproducir de forma manual. \n");
	manual = getchar();
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	int *MatrizAux_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(filas*columnas * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, filas*columnas * sizeof(int));
	hipMalloc((void**)&MatrizAux_d, filas*columnas * sizeof(int));
	//Inicializaci�n de matriz.
	int contadorSemillas = 0;
	for (int i = 0; i < filas*columnas; i++)
	{

		if (rand() % 100 < 25 && contadorSemillas < 9) //Solo puede haber un maximo de 9 semillas iniciales. Hay una posibilidad del 25% de que la posicion sea semilla.
		{
			MatrizA[i] = 1;
			contadorSemillas++;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	dim3 nBloques(1, 1);
	dim3 hilosBloque((columnas + nBloques.x - 1) / nBloques.x, (filas + nBloques.y - 1) / nBloques.y);
	caracter = getchar();
	while (caracter != 'p')
	{
		//Representaci�n de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < filas; i++)
		{
			for (int j = 0; j < columnas; j++)
			{
				printf("%d ", MatrizA[j + i * columnas]);
			}
			printf("\n");
		}
		//Env�o de datos al device.
		hipMemcpy(MatrizA_d, MatrizA, filas*columnas * sizeof(int), hipMemcpyHostToDevice);
		//Realizaci�n de la operaci�n.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, MatrizAux_d, filas, columnas);
		//Env�o de datos al host.
		hipMemcpy(MatrizA_d, MatrizAux_d, filas*columnas * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(MatrizA, MatrizA_d, filas*columnas * sizeof(int), hipMemcpyDeviceToHost);
		//Si esta en modo manual se espera a que el usuario pulse alguna tecla para continuar (Si es 'p' se para la ejecucion)
		if (manual == 'm') {
			caracter = getchar();
		}
		generacion += 1;
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	hipFree(MatrizAux_d);
	free(MatrizA);

}
