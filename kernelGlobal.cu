#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

__device__ int comprobarVecinos(int *a, int idCelula, int ladoMatriz, int largoMatriz)
{
	int idHilo = idCelula;
	int contador = 0;
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior izquierda de la matriz.
	if (idCelula == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina superior derecha de la matriz.
	else if (idHilo == ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior izquierda de la matriz.
	else if (idHilo == (ladoMatriz *  largoMatriz - ladoMatriz))
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo + 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en la esquina inferior derecha de la matriz.
	else if (idHilo == ladoMatriz * largoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[3] = { a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 3; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado izquierdo (pero no en la esquina) de la matriz.
	else if (idHilo % ladoMatriz == 0)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra en el lado derecho (pero no en la esquina) de la matriz.
	else if (idHilo % ladoMatriz == ladoMatriz - 1)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra abajo (pero no en la esquina) de la matriz.
	else if (idHilo > ladoMatriz * (largoMatriz - 1) && idHilo < ladoMatriz * largoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz + 1], a[idHilo - ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Comprobamos si el hilo que ha llamado al kernel se encuentra arriba (pero no en la esquina) de la matriz.
	else if (idHilo < ladoMatriz)
	{
		//Creamos array con vecinos de la celula
		int vecinos[5] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 5; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	//Hilo esta en el medio
	else
	{
		//Creamos array con vecinos de la celula
		int vecinos[8] = { a[idHilo + 1], a[idHilo - 1], a[idHilo + ladoMatriz], a[idHilo - ladoMatriz], a[idHilo - ladoMatriz - 1], a[idHilo - ladoMatriz + 1], a[idHilo + ladoMatriz + 1], a[idHilo + ladoMatriz - 1] };
		for (int i = 0; i < 8; i++)
		{
			if (vecinos[i] == 1)
			{
				contador += 1;
			}
		}
	}
	return contador;
}
__device__ void cambiarEstado(int *a, int *aux, int idCelula, int ladoMatriz, int largoMatriz)
{
	int idHilo = idCelula;
	int contador = comprobarVecinos(a, idHilo, ladoMatriz, largoMatriz);
	//La celula esta viva 
	if (a[idHilo] == 1 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 celulas vivas o mas de 3
		//Matamos la celula
		aux[idHilo] = 0;
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && contador == 3)
	{
		//Hay 3 celulas vivas alrededor
		//La celula nace
		aux[idHilo] = 1;
	}
	//La celula esta viva
	else if (a[idHilo] == 1 && (contador == 2 || contador == 3))
	{
		//Hay 2 o 3 c�lulas alrededor
		//La c�lula se mantiene viva
		aux[idHilo] = 1;
	}
	//La celula esta muerta
	else if (a[idHilo] == 0 && (contador < 2 || contador > 3))
	{
		//Hay menos de 2 o m�s de 3 c�lulas vivas alrededor
		//La c�lula se mantiene muerta
		aux[idHilo] = 0;
	}
}
//Funci�n de comprobaci�n a realizar por el kernel
__global__ void llamadaCelula(int *a, int *aux, int ladoMatriz, int largoMatriz)
{
	int idFila = threadIdx.x;
	int idColumna = threadIdx.y;
	int idHilo = idColumna + idFila * blockDim.x;
	cambiarEstado(a, aux, idHilo, ladoMatriz, largoMatriz);
	__syncthreads();
}

int main(int argc, char** argv)
{
	int ladoMatriz = 0;
	int largoMatriz = 0;
	char modo = ' ';
	char caracter = ' ';
	int generacion = 0;
	printf("Introduzca el metodo de ejecucion (m)anual o (a)utomatica. \n");
	modo = getchar();
	printf("Introduzca el ancho de la matriz. \n");
	scanf("%d", &ladoMatriz);
	getchar();
	printf("Introduzca el alto de la matriz. \n");
	scanf("%d", &largoMatriz);
	//Declaraciones de variables.
	int *MatrizA, *MatrizA_d;
	int *MatrizAux, *MatrizAux_d;
	//Reserva de memoria en el host.
	MatrizA = (int*)malloc(ladoMatriz*largoMatriz * sizeof(int));
	MatrizAux = (int*)malloc(ladoMatriz*largoMatriz * sizeof(int));
	//Reserva de memoria en el device.
	hipMalloc((void**)&MatrizA_d, ladoMatriz*largoMatriz * sizeof(int));
	hipMalloc((void**)&MatrizAux_d, ladoMatriz*largoMatriz * sizeof(int));
	//Inicializaci�n de matriz.
	int contadorSemillas = 0;
	for (int i = 0; i < ladoMatriz * largoMatriz; i++)
	{
		
		if (rand() % 2) //Hay una posibilidad del 50% de que la posicion sea una celula viva.
		{
			MatrizA[i] = 1;
		}
		else
		{
			MatrizA[i] = 0;
		}
	}
	//Inicializaci�n de l matriz auxiliar
	for (int i = 0; i < ladoMatriz * largoMatriz; i++)
	{
		MatrizAux[i] = 0;
	}
	dim3 nBloques(1, 1);
	dim3 hilosBloque(ladoMatriz, largoMatriz);
	hipMemcpy(MatrizAux_d, MatrizAux, ladoMatriz * largoMatriz * sizeof(int), hipMemcpyHostToDevice);
	caracter = getchar();
	while (caracter != 'p')
	{
		//Representaci�n de los resultados.
		printf("Matriz A en generacion %d:\n", generacion);
		for (int i = 0; i < largoMatriz; i++)
		{
			for (int j = 0; j < ladoMatriz; j++)
			{
				printf("%d ", MatrizA[j + i * ladoMatriz]);
			}
			printf("\n");
		}
		//Env�o de datos al device.
		hipMemcpy(MatrizA_d, MatrizA, ladoMatriz * largoMatriz * sizeof(int), hipMemcpyHostToDevice);
		//Realizaci�n de la operaci�n.
		llamadaCelula << <nBloques, hilosBloque >> > (MatrizA_d, MatrizAux_d, ladoMatriz, largoMatriz);
		//Env�o de datos al host.
		MatrizA_d = MatrizAux_d;
		hipMemcpy(MatrizA, MatrizA_d, ladoMatriz * largoMatriz * sizeof(int), hipMemcpyDeviceToHost);
		if (modo == 'm')
		{
			caracter = getchar();
		}
		else
		{
			Sleep(1000);
		}
		generacion += 1;
	}
	//Liberaci�n del espacio usado por los punteros.
	hipFree(MatrizA_d);
	hipFree(MatrizAux_d);
	free(MatrizA);
	free(MatrizAux);
}